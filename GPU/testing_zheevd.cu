#include "hip/hip_runtime.h"
#include "../Headers/mersenne_twister.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hip/hip_runtime_api.h>
#include "magma_v2.h"
#include "magma_lapack.h"

__global__ void SetRandomMatrix(int Dmat, magmaDoubleComplex* mat, hiprandStateMtgp32_t* MTGPStates_d, int nBlock) {
  int i = blockIdx.x*blockDim.x +threadIdx.x;
  int j = blockIdx.y*blockDim.y +threadIdx.y;
  //int blockId = blockIdx.x +nBlock*blockIdx.y;

  if( i<Dmat && j<Dmat ){
    double rand1 = hiprand_normal_double(&MTGPStates_d[0]) /sqrt(2.0);
    double rand2 = hiprand_normal_double(&MTGPStates_d[0]) /sqrt(2.0);
    //printf("%+.4lf, %+.4lf\n", rand1, rand2);

    if(i == j) mat[i +Dmat*j] = MAGMA_Z_MAKE(sqrt(2.0)*rand1,0);
    else if(i < j) {
      mat[i +Dmat*j] = MAGMA_Z_MAKE(rand1,rand2);
      mat[j +Dmat*i] = MAGMA_Z_CONJ(mat[i +Dmat*j]);
    }
  }
}

int main(int argc, char **argv) {
    if(argc != 2) {
      fprintf(stderr, "Usage: 1.This 2.Dmat\n");
      exit(1);
    }

    const int Dmat = atoi(argv[1]);
    int nThread = (int)sqrt(1024);
    int nBlock = (int)Dmat/nThread;
    if( Dmat%nThread != 0 ) nBlock += 1;
    printf("nBlock=%d, nThread=%d *%d=%d\n", nBlock, nThread, nThread, nThread*nThread);
  
    dim3 dimGrid(nBlock, nBlock, 1);
    dim3 dimBlock(nThread, nThread, 1);

    const unsigned long long seed = 0;
    hiprandStateMtgp32 *MTGPStates_d;
    mtgp32_kernel_params *KernelParams_d;
    //hiprandGenerator_t generator;
    //hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
    //hiprandSetPseudoRandomGeneratorSeed(generator, seed);

    void (*po)(int, magmaDoubleComplex*, hiprandStateMtgp32_t*, int);
    po = SetRandomMatrix;
    struct hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(po));
    printf("constSizeBytes     = %d\n", attr.constSizeBytes);
    printf("maxThreadsPerBlock = %d\n", attr.maxThreadsPerBlock);
    
      
    magma_init();
    magmaDoubleComplex *mat_d, *EigenVectors_d, *temp_d;
    double *w, *rwork;
    magmaDoubleComplex *wA, *work;
    magma_int_t lwork;
    magma_int_t lrwork = 1+5*Dmat+2*Dmat*Dmat;
    magma_int_t *iwork;
    magma_int_t liwork = 3+5*Dmat;
    magma_int_t info;
    magma_int_t nb = magma_get_zhetrd_nb(Dmat);

    struct timespec start_time, end_time;
    clock_t start, end;

    int temp1 = Dmat +Dmat*nb;
    int temp2 = 2*Dmat+Dmat*Dmat;
    if(temp1 < temp2) lwork = temp2;
    else lwork = temp1;

    magma_queue_t queue=NULL;
    magma_int_t dev = 0;
    magma_queue_create( dev, &queue );
    
    magma_zmalloc( &mat_d, Dmat*Dmat );
    magma_zmalloc( &EigenVectors_d, Dmat*Dmat );
    magma_zmalloc( &temp_d, Dmat*Dmat );

    magma_dmalloc_cpu( &w, Dmat );
    magma_zmalloc_cpu( &wA, Dmat*Dmat );
    magma_zmalloc_cpu( &work, lwork );
    magma_dmalloc_cpu( &rwork, lrwork );
    magma_imalloc_cpu( &iwork, liwork );
    
    hipMalloc( (void**)&MTGPStates_d, nBlock*nBlock *sizeof(hiprandStateMtgp32) );
    hipMalloc( (void**)&KernelParams_d, sizeof(mtgp32_kernel_params) );
    hipMemset( MTGPStates_d, 0, nBlock*nBlock *sizeof(hiprandStateMtgp32) );

    hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, KernelParams_d);
    hiprandMakeMTGP32KernelState(MTGPStates_d, mtgp32dc_params_fast_11213, KernelParams_d, 1, seed);
    SetRandomMatrix<<<dimGrid,dimBlock>>>(Dmat, mat_d, MTGPStates_d, nBlock);
    //magma_zprint_gpu(Dmat, Dmat, mat_d, Dmat, queue); 
    magma_zcopymatrix(Dmat, Dmat, mat_d, Dmat, EigenVectors_d, Dmat, queue);

    start = clock();
    clock_gettime(CLOCK_REALTIME, &start_time);
    magma_zheevd_gpu(MagmaVec, MagmaUpper, Dmat, EigenVectors_d, Dmat, w, wA, Dmat, work, lwork, rwork, lrwork, iwork, liwork, &info);
    
    magmaDoubleComplex alpha = MAGMA_Z_MAKE(1,0);
    magmaDoubleComplex beta  = MAGMA_Z_MAKE(0,0);
    magma_zhemm(MagmaLeft, MagmaUpper, Dmat, Dmat, alpha, mat_d, Dmat, EigenVectors_d, Dmat, beta, temp_d, Dmat, queue);
    //magma_zgemm(MagmaConjTrans, MagmaNoTrans, Dmat, Dmat, Dmat, alpha, EigenVectors_d, Dmat, temp_d, Dmat, beta, mat_d, Dmat, queue);
    /*
    magma_zprint_gpu(Dmat, Dmat, mat_d, Dmat, queue);
    magma_dprint(1, Dmat, w, 1);
    */

    end = clock();
    clock_gettime(CLOCK_REALTIME, &end_time);

    unsigned int sec;
    int nsec;
    double d_sec;
    sec = end_time.tv_sec - start_time.tv_sec;
    nsec = end_time.tv_nsec - start_time.tv_nsec;
    d_sec = (double)sec +(double)nsec / (1000 * 1000 * 1000);
    fprintf(stderr, "%d %f %f\n", Dmat, (double)(end-start)/CLOCKS_PER_SEC, d_sec);
    
    magma_queue_destroy(queue);
    magma_free(mat_d);
    magma_free(EigenVectors_d);
    magma_free(temp_d);

    magma_free_cpu(w);
    magma_free_cpu(wA);
    magma_free_cpu(work);
    magma_free_cpu(rwork);
    magma_free_cpu(iwork);

    hipFree(MTGPStates_d);
    hipFree(KernelParams_d);

    magma_finalize();
    //hiprandDestroyGenerator(generator);
    
    return 0;
}
