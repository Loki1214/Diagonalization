#include "hip/hip_runtime.h"
#include "../Headers/mersenne_twister.h"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hipblas.h>
//#include <hip/hip_runtime_api.h>
#include "magma_v2.h"
#include "magma_lapack.h"

int main(int argc, char **argv) {
    if(argc != 2) {
      fprintf(stderr, "Usage: 1.This 2.Dmat\n");
      exit(1);
    }
    const int seed = 0;
    init_genrand(seed);

    magma_init();
    const int Dmat = atoi(argv[1]);
    double *mat_h, *mat_d, *EigenVectors_d, *temp_d;
    double *w, *wA, *work;
    magma_int_t lwork;
    int *iwork;
    magma_int_t liwork = 3+5*Dmat;
    magma_int_t info;
    magma_int_t nb = magma_get_dsytrd_nb(Dmat);

    struct timespec start_time, end_time;
    clock_t start, end;

    int temp1 = 2*Dmat +Dmat*nb;
    int temp2 = 1+6*Dmat+2*Dmat*Dmat;
    if(temp1 < temp2) lwork = temp2;
    else lwork = temp1;

    magma_queue_t queue=NULL;
    magma_int_t dev = 0;
    magma_queue_create( dev, &queue );

    magma_dmalloc_pinned( &mat_h, Dmat*Dmat );
    magma_dmalloc( &mat_d, Dmat*Dmat );
    magma_dmalloc( &EigenVectors_d, Dmat*Dmat );
    magma_dmalloc( &temp_d, Dmat*Dmat );
    magma_dmalloc_cpu( &w, Dmat );
    magma_dmalloc_cpu( &wA, Dmat*Dmat );
    magma_dmalloc_cpu( &work, lwork );
    magma_imalloc_cpu( &iwork, liwork );

    for(int i=0;i < Dmat; ++i) {
      mat_h[i+Dmat*i] = genrand_real3();
      for(int j=0;j < i; ++j) {
	mat_h[i+Dmat*j] = genrand_real3();
	mat_h[j+Dmat*i] = mat_h[i+Dmat*j];
      }
    }
    //magma_dprint(Dmat, Dmat, mat_h, Dmat);

    magma_dsetmatrix(Dmat, Dmat, mat_h, Dmat, mat_d, Dmat, queue);
    magma_dcopymatrix(Dmat, Dmat, mat_d, Dmat, EigenVectors_d, Dmat, queue);

    start = clock();
    clock_gettime(CLOCK_REALTIME, &start_time);
    magma_dsyevd_gpu(MagmaVec, MagmaUpper, Dmat, EigenVectors_d, Dmat, w, wA, Dmat, work, lwork, iwork, liwork, &info);
    end = clock();
    clock_gettime(CLOCK_REALTIME, &end_time);

    unsigned int sec;
    int nsec;
    double d_sec;
    sec = end_time.tv_sec - start_time.tv_sec;
    nsec = end_time.tv_nsec - start_time.tv_nsec;
    d_sec = (double)sec +(double)nsec / (1000 * 1000 * 1000);
    fprintf(stdout, "%d %f %f\n", Dmat, (double)(end-start)/CLOCKS_PER_SEC, d_sec);

//    magma_dsymm(MagmaLeft, MagmaUpper, Dmat, Dmat, 1, mat_d, Dmat, EigenVectors_d, Dmat, 0, temp_d, Dmat, queue);
//    magma_dgemm(MagmaTrans, MagmaNoTrans, Dmat, Dmat, Dmat, 1, EigenVectors_d, Dmat, temp_d, Dmat, 0, mat_d, Dmat, queue);
//    magma_dprint_gpu(Dmat, Dmat, mat_d, Dmat, queue);

    magma_queue_destroy(queue);
    //magma_free_cpu(mat_h);
    //magma_free(mat_d);
    //magma_free_cpu(w);
    //magma_free_cpu(wA);
    //magma_free_cpu(work);
    //magma_free_cpu(iwork);
    return 0;
}
